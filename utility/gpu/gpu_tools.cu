// Copyright 2018 The Beam Team
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//    http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "gpu_tools.h"

#include "hip/hip_runtime.h"

namespace beam
{
bool HasSupportedCard()
{
    int device;
    hipError_t code = hipGetDevice(&device);
    if (code == hipSuccess)
        return true;

    return false;
}
}